#define TORCH_ASSERT_ONLY_METHOD_OPERATORS
#include <ATen/Dispatch.h>
#include <ATen/core/Tensor.h>
#include <ATen/cuda/HIPContext.h>
#include <ATen/cuda/nvrtc_stub/ATenNVRTC.h>

// Determine if the architecture supports rowwise scaled mm
// Currenlty failing on windows with: https://github.com/NVIDIA/cutlass/issues/1571
#if !defined(USE_ROCM) && !defined(_WIN32) && defined(CUDA_VERSION) && CUDA_VERSION >= 12000

#define BUILD_ROWWISE_FP8_KERNEL
#endif

#if defined(BUILD_ROWWISE_FP8_KERNEL)

// We are going to override the cuTensorMapEncodeTiled driver api with our lazy loader
static hipError_t CUDAAPI nvrtc_cuTensorMapEncodeTiled(
    CUtensorMap* tensorMap,
    CUtensorMapDataType tensorDataType,
    cuuint32_t tensorRank,
    void* globalAddress,
    const cuuint64_t* globalDim,
    const cuuint64_t* globalStrides,
    const cuuint32_t* boxDim,
    const cuuint32_t* elementStrides,
    CUtensorMapInterleave interleave,
    CUtensorMapSwizzle swizzle,
    CUtensorMapL2promotion l2Promotion,
    CUtensorMapFloatOOBfill oobFill) {
  return at::globalContext().getNVRTC().cuTensorMapEncodeTiled(
      tensorMap,
      tensorDataType,
      tensorRank,
      globalAddress,
      globalDim,
      globalStrides,
      boxDim,
      elementStrides,
      interleave,
      swizzle,
      l2Promotion,
      oobFill);
}


#include <cutlass/core_io.h>
#include <cutlass/cutlass.h>
#include <cutlass/gemm/device/gemm.h>
#include <cutlass/half.h>
#include <cutlass/numeric_types.h>
#include <cutlass/trace.h>
#include <cutlass/util/host_tensor.h>

// Rename the global function symbol
#define cuTensorMapEncodeTiled nvrtc_cuTensorMapEncodeTiled
#include <cute/tensor.hpp>
#undef cuTensorMapEncodeTiled
// Set everything back to normal

#include <cutlass/gemm/collective/collective_builder.hpp>
#include <cutlass/gemm/device/gemm_universal_adapter.h>
#include <cutlass/epilogue/collective/collective_builder.hpp>

#include <cute/atom/mma_atom.hpp>
#include <cutlass/gemm/dispatch_policy.hpp>
#include <cutlass/gemm/kernel/gemm_universal.hpp>
#include <cutlass/util/packed_stride.hpp>


namespace {

using DtypeScale = float;
using DtypeAccum = float;
using DtypeEpilogue = float;
using DtypeOutput = cutlass::bfloat16_t;

using Multiply = cutlass::epilogue::fusion::Sm90Compute<
    cutlass::multiplies,
    DtypeEpilogue,
    DtypeEpilogue,
    cutlass::FloatRoundStyle::round_to_nearest>;

using Add = cutlass::epilogue::fusion::Sm90Compute<
    cutlass::plus,
    DtypeEpilogue,
    DtypeEpilogue,
    cutlass::FloatRoundStyle::round_to_nearest>;

using Cast = cutlass::epilogue::fusion::Sm90Compute<
    cutlass::epilogue::thread::Identity,
    DtypeOutput,
    DtypeEpilogue,
    cutlass::FloatRoundStyle::round_to_nearest>;

template <bool PingPong, bool FastAccum>
struct Schedule;

template <>
struct Schedule</*PingPong=*/false, /*FastAccum=*/false> {
  using type = cutlass::gemm::KernelTmaWarpSpecialized;
};

template <>
struct Schedule</*PingPong=*/true, /*FastAccum=*/false> {
  using type = cutlass::gemm::KernelTmaWarpSpecializedPingpong;
};

template <>
struct Schedule</*PingPong=*/false, /*FastAccum=*/true> {
  using type = cutlass::gemm::KernelTmaWarpSpecializedFP8FastAccum;
};

template <>
struct Schedule</*PingPong=*/true, /*FastAccum=*/true> {
  using type = cutlass::gemm::KernelTmaWarpSpecializedPingpongFP8FastAccum;
};

// Cutlass rowwise kernel
template <
    typename TileShape,
    typename ClusterShape,
    typename PingPong,
    typename FastAccum,
    typename DtypeA,
    typename DtypeB,
    typename DtypeBias>
void f8f8bf16_rowwise_impl(
    at::Tensor XQ, // FP8
    at::Tensor WQ, // FP8
    at::Tensor x_scale,
    at::Tensor w_scale,
    std::optional<at::Tensor> bias,
    at::Tensor out) {
  int M = XQ.size(0);
  int N = WQ.size(1);
  int K = XQ.size(1);

  // Workaround for https://github.com/pytorch/pytorch/issues/133334.
  if (M % 256 > 0) {
    int padded_M = ((M - 1) / 256 + 1) * 256;
    at::Tensor padded_x_scale = x_scale.new_empty({padded_M, 1});
    padded_x_scale.slice(/*dim=*/0, /*start=*/0, /*end=*/M)
        .copy_(std::move(x_scale));
    x_scale = std::move(padded_x_scale);
  }

  using LayoutInputA = cutlass::layout::RowMajor;
  constexpr int AlignmentInputA = 16 / sizeof(DtypeA);

  using LayoutInputB = cutlass::layout::ColumnMajor;
  constexpr int AlignmentInputB = 16 / sizeof(DtypeB);

  using LayoutOutput = cutlass::layout::RowMajor;
  constexpr int AlignmentOutput = 16 / sizeof(DtypeOutput);

  // Tag indicating the minimum SM that supports the intended feature
  using ArchTag = cutlass::arch::Sm90;
  using OperatorClass = cutlass::arch::OpClassTensorOp;

  // Implement rowwise scaling epilogue.
  constexpr int ColBroadcastStages = 0;
  constexpr int RowBroadcastStages = PingPong::value ? 2 : 1;

  using XScale = cutlass::epilogue::fusion::
      Sm90ColBroadcast<ColBroadcastStages, TileShape, DtypeScale>;

  using WScale = cutlass::epilogue::fusion::
      Sm90RowBroadcast<RowBroadcastStages, TileShape, DtypeScale>;

  using Bias = cutlass::epilogue::fusion::
      Sm90RowBroadcast<RowBroadcastStages, TileShape, DtypeBias>;

  using Accum = cutlass::epilogue::fusion::Sm90AccFetch;

  using EpilogueEVT = cutlass::epilogue::fusion::Sm90EVT<
      Cast,
      cutlass::epilogue::fusion::Sm90EVT<
          Add,
          Bias,
          cutlass::epilogue::fusion::Sm90EVT<
              Multiply,
              XScale,
              cutlass::epilogue::fusion::Sm90EVT<Multiply, WScale, Accum>>>>;

  using CollectiveEpilogue =
      typename cutlass::epilogue::collective::CollectiveBuilder<
          ArchTag,
          OperatorClass,
          TileShape,
          ClusterShape,
          cutlass::epilogue::collective::EpilogueTileAuto,
          DtypeAccum,
          DtypeEpilogue,
          DtypeOutput,
          LayoutOutput,
          AlignmentOutput,
          DtypeOutput,
          LayoutOutput,
          AlignmentOutput,
          cutlass::epilogue::TmaWarpSpecialized,
          EpilogueEVT>::CollectiveOp;

  using CollectiveMainloop =
      typename cutlass::gemm::collective::CollectiveBuilder<
          ArchTag,
          OperatorClass,
          DtypeA,
          LayoutInputA,
          AlignmentInputA,
          DtypeB,
          LayoutInputB,
          AlignmentInputB,
          DtypeAccum,
          TileShape,
          ClusterShape,
          cutlass::gemm::collective::StageCountAutoCarveout<static_cast<int>(
              sizeof(typename CollectiveEpilogue::SharedStorage))>,
          typename Schedule<PingPong::value, FastAccum::value>::type>::
          CollectiveOp;

  using GemmKernel = cutlass::gemm::kernel::GemmUniversal<
      cute::Shape<int, int, int>,
      CollectiveMainloop,
      CollectiveEpilogue>;

  using Gemm = cutlass::gemm::device::GemmUniversalAdapter<GemmKernel>;

  using StrideInputA = typename Gemm::GemmKernel::StrideA;
  using StrideInputB = typename Gemm::GemmKernel::StrideB;
  using StrideOutput = typename Gemm::GemmKernel::StrideC;

  StrideInputA stride_a = cutlass::make_cute_packed_stride(
      StrideInputA{}, cute::make_shape(M, static_cast<int>(XQ.stride(0)), 1));
  StrideInputB stride_b = cutlass::make_cute_packed_stride(
      StrideInputB{}, cute::make_shape(N, static_cast<int>(WQ.stride(1)), 1));
  StrideOutput stride_output = cutlass::make_cute_packed_stride(
      StrideOutput{}, cute::make_shape(M, static_cast<int>(out.stride(0)), 1));

  typename Gemm::Arguments arguments{
      cutlass::gemm::GemmUniversalMode::kGemm,
      {M, N, K},
      {reinterpret_cast<DtypeA*>(XQ.data_ptr()),
       stride_a,
       reinterpret_cast<DtypeB*>(WQ.data_ptr()),
       stride_b},
      {{{{bias.has_value() ? reinterpret_cast<DtypeBias*>(bias->data_ptr())
                           : nullptr},
         {{reinterpret_cast<DtypeScale*>(x_scale.data_ptr())},
          {{reinterpret_cast<DtypeScale*>(w_scale.data_ptr())}}}}},
       reinterpret_cast<DtypeOutput*>(out.data_ptr()),
       stride_output,
       reinterpret_cast<DtypeOutput*>(out.data_ptr()),
       stride_output}};

  Gemm gemm;

  // Using the arguments, query for extra workspace required for matrix
  // multiplication computation
  size_t workspace_size = Gemm::get_workspace_size(arguments);

  // Allocate workspace memory
  auto workspace = XQ.new_empty(
      {static_cast<int64_t>(workspace_size)},
      at::TensorOptions().dtype(at::kByte));

  // Check the problem size is supported or not
  cutlass::Status status = gemm.can_implement(arguments);
  if (status != cutlass::Status::kSuccess) {
    throw std::runtime_error("cutlass cannot implement");
  }

  // Initialize CUTLASS kernel with arguments and workspace pointer
  status = gemm.initialize(arguments, workspace.data_ptr());
  if (status != cutlass::Status::kSuccess) {
    throw std::runtime_error("cutlass cannot initialize");
  }

  status = gemm(at::cuda::getCurrentCUDAStream());
  if (status != cutlass::Status::kSuccess) {
    throw std::runtime_error(
        std::string("cutlass cannot run") +
        cutlass::cutlassGetStatusString(status));
  }
  C10_CUDA_KERNEL_LAUNCH_CHECK();
}

// FP8 Rowwise Cutlass kernel dispatch.
enum class KernelMode { Small, Large, Default };

KernelMode get_kernel_mode(at::Tensor XQ, at::Tensor WQ) {
  auto M = XQ.size(0);
  auto K = XQ.size(1);
  auto N = WQ.size(0);
  // Use a large kernel if at least two shapes are large....
  bool use_large_kernel =
      ((M >= 2048 && K >= 2048) || (M >= 2048 && N >= 2048) ||
       (K >= 2048 && N >= 2048));
  if (M <= 128 || N <= 128) {
    return KernelMode::Small;
  } else if (use_large_kernel) {
    return KernelMode::Large;
  } else {
    return KernelMode::Default;
  }
}

template <typename... Types>
void dispatch_fp8_rowwise_kernel_on_tile_size(
    at::Tensor XQ,
    at::Tensor WQ,
    at::Tensor x_scale,
    at::Tensor w_scale,
    std::optional<at::Tensor> bias,
    at::Tensor out) {
  KernelMode kernel = get_kernel_mode(XQ, WQ);
  if (kernel == KernelMode::Small) {
    return f8f8bf16_rowwise_impl<
        /*TileShape=*/cute::Shape<cute::_64, cute::_128, cute::_128>,
        /*ClusterShape=*/cute::Shape<cute::_2, cute::_1, cute::_1>,
        /*PingPong=*/std::false_type,
        Types...>(XQ, WQ, x_scale, w_scale, bias, out);
  } else if (kernel == KernelMode::Large) {
    return f8f8bf16_rowwise_impl<
        /*TileShape=*/cute::Shape<cute::_128, cute::_128, cute::_128>,
        /*ClusterShape=*/cute::Shape<cute::_2, cute::_1, cute::_1>,
        /*PingPong=*/std::true_type,
        Types...>(XQ, WQ, x_scale, w_scale, bias, out);
  } else {
    return f8f8bf16_rowwise_impl<
        /*TileShape=*/cute::Shape<cute::_128, cute::_128, cute::_128>,
        /*ClusterShape=*/cute::Shape<cute::_1, cute::_2, cute::_1>,
        /*PingPong=*/std::false_type,
        Types...>(XQ, WQ, x_scale, w_scale, bias, out);
  }
}

template <typename... Types>
void dispatch_fp8_rowwise_kernel_on_fast_accum(
    at::Tensor XQ,
    at::Tensor WQ,
    at::Tensor x_scale,
    at::Tensor w_scale,
    std::optional<at::Tensor> bias,
    bool use_fast_accum,
    at::Tensor out) {
  if (use_fast_accum) {
    dispatch_fp8_rowwise_kernel_on_tile_size<std::true_type, Types...>(
        XQ, WQ, x_scale, w_scale, bias, out);
  } else {
    dispatch_fp8_rowwise_kernel_on_tile_size<std::false_type, Types...>(
        XQ, WQ, x_scale, w_scale, bias, out);
  }
}

template <typename... Types>
void dispatch_fp8_rowwise_kernel_on_input_dtypes(
    at::Tensor XQ,
    at::Tensor WQ,
    at::Tensor x_scale,
    at::Tensor w_scale,
    std::optional<at::Tensor> bias,
    bool use_fast_accum,
    at::Tensor out) {
  if (XQ.dtype() == at::kFloat8_e5m2) {
    dispatch_fp8_rowwise_kernel_on_fast_accum<
        cutlass::float_e5m2_t,
        cutlass::float_e4m3_t,
        Types...>(XQ, WQ, x_scale, w_scale, bias, use_fast_accum, out);
  } else {
    dispatch_fp8_rowwise_kernel_on_fast_accum<
        cutlass::float_e4m3_t,
        cutlass::float_e4m3_t,
        Types...>(XQ, WQ, x_scale, w_scale, bias, use_fast_accum, out);
  }
}

void dispatch_fp8_rowwise_kernel_on_bias_dtype(
    at::Tensor XQ,
    at::Tensor WQ,
    at::Tensor x_scale,
    at::Tensor w_scale,
    std::optional<at::Tensor> bias,
    bool use_fast_accum,
    at::Tensor out) {
  if (bias.has_value() && bias->dtype() == at::kBFloat16) {
    dispatch_fp8_rowwise_kernel_on_input_dtypes<cutlass::bfloat16_t>(
        XQ, WQ, x_scale, w_scale, bias, use_fast_accum, out);
  } else {
    dispatch_fp8_rowwise_kernel_on_input_dtypes<float>(
        XQ, WQ, x_scale, w_scale, bias, use_fast_accum, out);
  }
}

void check_inputs(
    const at::Tensor& a,
    const at::Tensor& b,
    const at::Tensor& scale_a,
    const at::Tensor& scale_b,
    const std::optional<at::Tensor>& bias,
    const at::Tensor& out) {
  TORCH_CHECK(a.is_cuda());
  TORCH_CHECK(a.device() == b.device());
  TORCH_CHECK(scale_a.device() == a.device());
  TORCH_CHECK(scale_b.device() == b.device());

  TORCH_CHECK(a.dtype() == at::kFloat8_e4m3fn || a.dtype() == at::kFloat8_e5m2);
  TORCH_CHECK(b.dtype() == at::kFloat8_e4m3fn);
  TORCH_CHECK(scale_a.dtype() == at::kFloat);
  TORCH_CHECK(scale_b.dtype() == at::kFloat);

  TORCH_CHECK(a.dim() == 2);
  TORCH_CHECK(b.dim() == 2);
  TORCH_CHECK(a.size(1) == b.size(0));
  TORCH_CHECK(scale_a.dim() == 2);
  TORCH_CHECK(scale_b.dim() == 2);
  TORCH_CHECK(scale_a.size(0) == a.size(0));
  TORCH_CHECK(scale_a.size(1) == 1);
  TORCH_CHECK(scale_b.size(0) == 1);
  TORCH_CHECK(scale_b.size(1) == b.size(1));

  TORCH_CHECK(a.stride(1) == 1);
  TORCH_CHECK(a.stride(0) >= a.size(1));
  TORCH_CHECK(b.stride(0) == 1);
  TORCH_CHECK(b.stride(1) >= b.size(0));
  TORCH_CHECK(scale_a.stride(0) == 1);
  TORCH_CHECK(scale_b.stride(1) == 1);

  if (bias.has_value()) {
    TORCH_CHECK(bias->device() == b.device());
    TORCH_CHECK(bias->dtype() == at::kFloat || bias->dtype() == at::kBFloat16);
    TORCH_CHECK(bias->dim() == 1);
    TORCH_CHECK(bias->size(0) == b.size(1));
    TORCH_CHECK(bias->stride(0) == 1);
  }

  TORCH_CHECK(out.device() == a.device());
  TORCH_CHECK(out.dtype() == at::kBFloat16);
  TORCH_CHECK(out.dim() == 2);
  TORCH_CHECK(out.size(0) == a.size(0));
  TORCH_CHECK(out.size(1) == b.size(1));
  TORCH_CHECK(out.stride(1) == 1);
  TORCH_CHECK(out.stride(0) >= out.size(1));
}

} // namespace

#endif // !defined(USE_ROCM)

namespace at::cuda::detail {
void f8f8bf16_rowwise(
    at::Tensor XQ, // FP8
    at::Tensor WQ, // FP8
    at::Tensor x_scale, // FP32
    at::Tensor w_scale, // FP32
    std::optional<at::Tensor> bias, // BF16
    bool use_fast_accum,
    at::Tensor& out) {
#if defined(BUILD_ROWWISE_FP8_KERNEL)
  check_inputs(XQ, WQ, x_scale, w_scale, bias, out);

  dispatch_fp8_rowwise_kernel_on_bias_dtype(
      XQ, WQ, x_scale, w_scale, bias, use_fast_accum, out);
#else // BUILD_ROWWISE_FP8_KERNEL
  TORCH_CHECK(
      false, "Rowwise scaling is not currenlty supported on your device");
#endif
}

} // namespace at::cuda::detail
